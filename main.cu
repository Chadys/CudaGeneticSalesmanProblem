#include "header.cuh"
#include "sort.cuh"
#include "solver.cuh"

int main() {
    // Init CUDA
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);


    // Init random cities
    float cpu_cities[N][2];
    for(int i = 0; i < N; ++i)
    {
        cpu_cities[i][0] = (float)rand() / RAND_MAX;
        cpu_cities[i][1] = (float)rand() / RAND_MAX;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(cities), &cpu_cities, sizeof(float) * N * 2);

    // Init gpu migrants
    Individu cpu_migrants[N];
    for(int i = 0; i < N; ++i)
    {
        cpu_migrants[i].score = -1;
        for (int j = 0; j < N; ++j)
        {
            cpu_migrants[i].path_indexes[j] = j;
        }
    }
    hipMemcpy(&gpu_migrants, cpu_migrants, sizeof(Individu) * N, hipMemcpyHostToDevice);

    // Init threads
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    int nb_threads = deviceProp.sharedMemPerBlock / sizeof(Individu);
    if(nb_threads > maxThreadsPerBlock)
        nb_threads = maxThreadsPerBlock;
    printf("Launching on %d threads\n", nb_threads);

    solve <<<1, nb_threads, nb_threads * sizeof(Individu)>>>();

    hipDeviceReset();
    return 0;
}
