#include "header.cuh"
#include "sort.cuh"
#include "solver.cuh"
#include <hip/hip_runtime_api.h>

__constant__ float cities[N_CITIES][2];

int getMaxNbThread(hipDeviceProp_t deviceProp){
    int quantity_in_each_thread = sizeof(Individu) + 10 * sizeof(int);
    int memory_available = deviceProp.sharedMemPerBlock - (N_CITIES * sizeof(bool) + (N_CITIES * sizeof(int)));


    int nb_threads = memory_available / quantity_in_each_thread;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    if(nb_threads > maxThreadsPerBlock)
        nb_threads = maxThreadsPerBlock;

    return nb_threads;
}

int main() {
    // Init CUDA
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Init random cities
    float cpu_cities[N_CITIES][2];
    for(int i = 0; i < N_CITIES; ++i) {
        cpu_cities[i][0] = (float)rand() / RAND_MAX;
        cpu_cities[i][1] = (float)rand() / RAND_MAX;
        //printf("(cpu) %f %f\n", cpu_cities[i][0], cpu_cities[i][1]);
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cities), cpu_cities, sizeof(float) * N_CITIES * 2));
    // Init gpu migrants
    Individu *gpu_migrants; // Les migrants ne sont pas en shared memory car ils sont partagés entre les blocs
    checkCudaErrors(hipMalloc(&gpu_migrants, sizeof(Individu) * N_ISLAND));

    // Init threads
    int nb_threads = getMaxNbThread(deviceProp);
    printf("Launching on %d threads\n", nb_threads);
    solve <<<N_ISLAND, nb_threads, (nb_threads * sizeof(Individu)) + (N_CITIES * sizeof(int)) + (N_CITIES * sizeof(bool))>>>(gpu_migrants);
    hipDeviceSynchronize();
    hipFree(gpu_migrants);
    hipDeviceReset();
    return 0;
}
