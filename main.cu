#include "header.cuh"
#include "sort.cuh"
#include "solver.cuh"
#include <hip/hip_runtime_api.h>

__constant__ float cities[N_CITIES][2];

int get_nb_max_thread(hipDeviceProp_t deviceProp){
    int quantity_in_each_thread = sizeof(Individu) + 10 * sizeof(int);
    int memory_available = deviceProp.sharedMemPerBlock - (N_CITIES * sizeof(bool) + (N_CITIES * sizeof(int)));


    int nb_threads = memory_available / quantity_in_each_thread;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    if(nb_threads > maxThreadsPerBlock)
        nb_threads = maxThreadsPerBlock;

    return nb_threads;
}

int main() {
    // Init CUDA
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Init random cities
    float cpu_cities[N_CITIES][2];
    for(int i = 0; i < N_CITIES; ++i) {
        cpu_cities[i][0] = (float)rand() / RAND_MAX;
        cpu_cities[i][1] = (float)rand() / RAND_MAX;
        //printf("(cpu) %f %f\n", cpu_cities[i][0], cpu_cities[i][1]);
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cities), cpu_cities, sizeof(float) * N_CITIES * 2));
    // Init gpu migrants
    Individu *gpu_migrants; // Migrants are not in shared memory because they need to be used by all bloc
    checkCudaErrors(hipMalloc(&gpu_migrants, sizeof(Individu) * N_ISLAND));

    // Init threads
    int nb_threads = get_nb_max_thread(deviceProp);
    printf("Launching on %d threads\n", nb_threads);
    solve <<<N_ISLAND, nb_threads, (nb_threads * sizeof(Individu)) + (N_CITIES * sizeof(int)) + (N_CITIES * sizeof(bool))>>>(gpu_migrants);
    hipDeviceSynchronize();
    hipFree(gpu_migrants);
    hipDeviceReset();
    return 0;
}
