#include "header.cuh"
#include "sort.cuh"
#include "solver.cuh"
#include <hip/hip_runtime_api.h>

__constant__ float cities[N_CITIES][2];

int getMaxNbThread(hipDeviceProp_t deviceProp){
    int quantity_in_each_thread = sizeof(Individu);
    int memory_available = deviceProp.sharedMemPerBlock; //TODO substract size of objects put in shared memory independently of thread number

    int nb_threads = memory_available / quantity_in_each_thread;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    if(nb_threads > maxThreadsPerBlock)
        nb_threads = maxThreadsPerBlock;

    return nb_threads;
}

int main() {
    // Init CUDA
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Init random cities
    float cpu_cities[N_CITIES][2];
    for(int i = 0; i < N_CITIES; ++i) {
        cpu_cities[i][0] = (float)rand() / RAND_MAX;
        cpu_cities[i][1] = (float)rand() / RAND_MAX;
        //printf("(cpu) %f %f\n", cpu_cities[i][0], cpu_cities[i][1]);
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cities), cpu_cities, sizeof(float) * N_CITIES * 2));
    // Init gpu migrants
    Individu *gpu_migrants;
    checkCudaErrors(hipMalloc(&gpu_migrants, sizeof(Individu) * N_ISLAND));

    // Init threads
    int nb_threads = getMaxNbThread(deviceProp);
    printf("Launching on %d threads\n", nb_threads);
    solve <<<N_ISLAND, nb_threads, nb_threads * sizeof(Individu)>>>(gpu_migrants);
    hipDeviceSynchronize();
    hipFree(gpu_migrants);
    hipDeviceReset();
    return 0;
}
