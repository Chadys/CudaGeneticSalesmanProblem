
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

#define N 10
#define N2 20

//TODO cities in const cache

__device__ void randomInit(int *individu, int *cities, hiprandState_t *state){
    bool used[N] = {false};
    for (int i = 0 ; i < N2 ; i+=2){
        int index = (int)(hiprand_uniform(state) * N);
        while (used[index])
            index = (index + 1) % N;
        used[index] = true;
        index *= 2;
        individu[i] = cities[index];
        individu[i+1] = cities[index+1];
    }
}

__global__ void solve(int *cities){
    extern __shared__ int population[];

    hiprandState_t state;
    hiprand_init(threadIdx.x, 0, 0, &state);

    randomInit((int *)(population + (threadIdx.x * N2)), cities, &state);

}

int main() {
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    int cities[N2] = {
            0, 2,
            1, 9,
            2, 14,
            4, 2,
            5, 7,
            8, 5,
            8, 12,
            11, 3,
            12, 11,
            13, 1
    }; //coordinate of all cities, x, y
    int *dC;
    int sizeVec = N2 * sizeof(int);

    hipMalloc(&dC, sizeVec);
    hipMemcpy(dC, cities, sizeVec, hipMemcpyHostToDevice);


    int nb_threads = (int)(deviceProp.sharedMemPerBlock / sizeof(int) / N2);
    if(nb_threads > maxThreadsPerBlock)
        nb_threads = maxThreadsPerBlock;
    printf("Launching on %d threads\n", nb_threads);

    solve <<<2, nb_threads, nb_threads * sizeof(int) * N2>>>(dC);
//    cudaMemcpy(C, dC, sizeVec, cudaMemcpyDeviceToHost);
    hipFree(dC);

    hipDeviceReset();
    return 0;
}
