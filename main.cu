
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>

#define N 10
#define N2 20

//TODO cities in const cache

__device__ void randomInit(int *individu, int *cities, hiprandState_t *state){
    bool used[N] = {false};
    for (int i = 0 ; i < N2 ; i+=2){
        int index = (int)(hiprand_uniform(state) * N);
        while (used[index])
            index ++;
        used[index] = true;
        index *= 2;
        individu[i] = cities[index];
        individu[i+1] = cities[index+1];
    }
}

__global__ void solve(int *cities){
    hiprandState_t state;
    hiprand_init(threadIdx.x, 0, 0, &state);
    int individu[N2];
    randomInit(individu, cities, &state);
}

int main() {
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    int cities[N2] = {
            0, 2,
            1, 9,
            2, 14,
            4, 2,
            5, 7,
            8, 5,
            8, 12,
            11, 3,
            12, 11,
            13, 1
    }; //coordinate of all cities, x, y
    int *dC;
    int sizeVec = N2 * sizeof(int);

    hipMalloc(&dC, sizeVec);
    hipMemcpy(dC, cities, sizeVec, hipMemcpyHostToDevice);

    printf("Launching on %d threads\n", maxThreadsPerBlock);
    solve <<<1, maxThreadsPerBlock>>>(dC);
//    cudaMemcpy(C, dC, sizeVec, cudaMemcpyDeviceToHost);
    hipFree(dC);

    return 0;
}
