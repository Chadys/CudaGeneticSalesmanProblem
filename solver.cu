#include "hip/hip_runtime.h"
#include "solver.cuh"

__device__ void update_score(Individu *individu) {
    double score = 0.f;
    int prevIndex = individu->pathIndexes[0];
    for(int i = 1; i < N_CITIES; i++) {
        int current_index = individu->pathIndexes[i];
//        if(threadIdx.x == 0) {
//            printf("%d %f %f\n", current_index, cities[current_index][0], cities[current_index][1]);
//        }
        score += pow(cities[current_index][0] - cities[prevIndex][0], 2) + pow(cities[current_index][1] - cities[prevIndex][1], 2);
        prevIndex = current_index;
    }
    individu->score = (float)score;
    //printf("%d : score = %f\n", threadIdx.x, (float)score);
}

__device__ bool is_gonna_die(hiprandState_t *state){
    float position = 1 - ((float)(threadIdx.x) / (blockDim.x - 1)); //first thread is 1.0, last is 0.0
    float powK = pow(position, PROBA_K);
    float probaToDie =  0.75f * powK;//(powk - (powk / (PROBA_K))) / PROBA_K;
    return hiprand_uniform(state) < probaToDie;
}

__device__ bool is_mutating(hiprandState_t *state){
    return hiprand_uniform(state) < PROBA_MUTATION;
}

__device__ void random_init(Individu *individu, hiprandState_t *state){
    bool used[N_CITIES] = {false};
    for (int i = 0 ; i < N_CITIES ; i++) {
        unsigned short index = (unsigned short)(hiprand_uniform(state) * N_CITIES);
        while (used[index])
            index = (unsigned short)((index + 1) % N_CITIES);
        used[index] = true;
        individu->pathIndexes[i] = index;
    }
}

__device__ Individu select_migrant(Individu *migrants, hiprandState_t *state) {
    unsigned short index = (unsigned short)(hiprand_uniform(state) * N_ISLAND);
    if (index == blockIdx.x)
        index = (unsigned short)((index + 1) % N_ISLAND);
    return migrants[index];
}

__device__ void select_mutation(hiprandState_t *state, unsigned short *mutation) {
    mutation[0] = (unsigned short)(hiprand_uniform(state) * N_CITIES);
    mutation[1] = (unsigned short)(hiprand_uniform(state) * N_CITIES);
    if (mutation[1] == mutation[0])
        mutation[1] = (unsigned short)((mutation[1] + 1) % N_CITIES);
}

__device__ void select_parents(hiprandState_t *state, int *parents, int numbersOfParents) {
    int current_parent = 0;
    while (current_parent < numbersOfParents) {
        for(int i = blockDim.x - 1; i >= 0; --i) {
            if(hiprand_uniform(state) < PROBA_SELECTION) {
                parents[current_parent++] = i;
                break;
            }
        }
    }
}

__device__ void mix_parents(Individu *population, hiprandState_t *state, int replacedIndex, int *parents,
                            int numbersOfParents) {
    int chunkSize = ceil((float)N_CITIES / numbersOfParents);
    int taken;
    for (int citiesCount = 0 ; citiesCount < N_CITIES ; citiesCount += taken) {
        int selected_parent = parents[hiprand(state) % numbersOfParents];//(chunkSize * 2)
        taken = hiprand(state) % (chunkSize * 2);
        if(citiesCount + taken > N_CITIES)
            taken = N_CITIES - citiesCount; // si on dépasse, on prend le reste
        for(int i = citiesCount; i < citiesCount + taken; ++i) {
            population[replacedIndex].pathIndexes[i] = population[selected_parent].pathIndexes[i];
        }
    }
}

__device__ void swap_cities(Individu *ind, unsigned short *citiesIndex){
    ind->pathIndexes[citiesIndex[0]] ^= ind->pathIndexes[citiesIndex[1]];
    ind->pathIndexes[citiesIndex[1]] ^= ind->pathIndexes[citiesIndex[0]];
    ind->pathIndexes[citiesIndex[0]] ^= ind->pathIndexes[citiesIndex[1]];
}

__device__ void print_path(Individu ind) {
    for(int i = 0; i < N_CITIES; i++) {
        printf("%2hu ", ind.pathIndexes[i]);
    }
    printf("\n");
}

__global__ void solve(Individu *migrants) {
    extern __shared__ Individu population[];

    hiprandState_t state;
    hiprand_init(threadIdx.x, 0, 0, &state);

    random_init(population + threadIdx.x, &state);
    update_score(population + threadIdx.x);

    if (threadIdx.x == 0) {
        //fill this block's migrant as soon as possible to be sure first migrant selection from another island won't get an uninitialized individual
        migrants[blockIdx.x] = population[0];
    }
    __syncthreads();
    merge_sort(population);


    // Main generation loop
    for(int i = 0; i < N_GENERATION ; i++) {
        __syncthreads();
        if (threadIdx.x == 0) {
            printf("GENERATION %d\n", i);
            migrants[blockIdx.x] = population[blockDim.x-1]; //export migrant
            population[0] = select_migrant(migrants, &state); //import migrant
        } else if(is_gonna_die(&state)) {
            int parents[3];
            select_parents(&state, parents, 3);
            printf("%d is dying. New parents : %d & %d & %d\n", threadIdx.x, parents[0], parents[1], parents[2]);
//            print_path(population[parents[0]]);
//            print_path(population[parents[1]]);
//            print_path(population[parents[2]]);
            mix_parents(population, &state, threadIdx.x, parents, 3);
            update_score(&population[threadIdx.x]);
//            print_path(population[threadIdx.x]);
        } else if(is_mutating(&state)) {
            printf("%d is mutating.\n", threadIdx.x);
            unsigned short citiesToBeExchanged[2];
            select_mutation(&state, citiesToBeExchanged);
            swap_cities(population + threadIdx.x, citiesToBeExchanged);
            update_score(&population[threadIdx.x]);
        }

        __syncthreads();
        merge_sort(population);
        //TODO replace with better specialized sort
    }

}