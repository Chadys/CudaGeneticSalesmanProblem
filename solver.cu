#include "hip/hip_runtime.h"
#include "solver.cuh"

extern __device__ Individu *gpu_migrants;

extern __constant__ float cities[N][2];


__device__ void updateScore(Individu *individu)
{
    double score = 0.f;
    int prev_index = individu->path_indexes[0];
    for(int i = 1; i < N; i++)
    {
        int current_index = individu->path_indexes[i];
        if(threadIdx.x == 0)
        {
            //printf("%d %f %f\n", current_index, cities[current_index][0], cities[current_index][1]);
        }
        score += powf(cities[current_index][0] - cities[prev_index][0], 2) + powf(cities[current_index][1] - cities[prev_index][1], 2);
        prev_index = current_index;
    }
    individu->score = (float)score;
    //printf("%d : score = %f\n", threadIdx.x, (float)score);
}

__device__ void randomInit(Individu *individu, hiprandState_t *state){
    bool used[N] = {false};
    for (int i = 0 ; i < N ; i++){
        int index = (int)(hiprand_uniform(state) * N);
        while (used[index])
            index = (index + 1) % N;
        used[index] = true;
        individu->path_indexes[i] = index;
    }
}

__global__ void solve(){
    extern __shared__ Individu population[];

    hiprandState_t state;
    hiprand_init(threadIdx.x, 0, 0, &state);

    randomInit(population + threadIdx.x, &state);
    updateScore(&population[threadIdx.x]);
    __syncthreads();
    if (threadIdx.x == 0) {
        for(int i = 0; i < blockDim.x; ++i)
        {
            printf("%d : %f\n", i, (population + i)->score);
        }
        /*
        for (int i = 0; i < N; i++) {
            printf("%d : %f\n", (population + threadIdx.x)->path_indexes[i]);
        }
         */
    }
}