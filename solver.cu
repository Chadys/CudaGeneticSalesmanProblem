#include "hip/hip_runtime.h"
#include "solver.cuh"

__device__ void updateScore(Individu *individu)
{
    double score = 0.f;
    int prev_index = individu->path_indexes[0];
    for(int i = 1; i < N; i++)
    {
        int current_index = individu->path_indexes[i];
        score += powf(cities[current_index][0] - cities[prev_index][0], 2) + powf(cities[current_index][1] - cities[prev_index][1], 2);
        prev_index = current_index;
    }
    individu->score = (float)score;
}

__device__ void randomInit(Individu *individu, hiprandState_t *state){
    bool used[N] = {false};
    for (int i = 0 ; i < N ; i++){
        int index = (int)(hiprand_uniform(state) * N);
        while (used[index])
            index = (index + 1) % N;
        used[index] = true;
        individu->path_indexes[i] = index;
    }
}

__global__ void solve(){
    extern __shared__ Individu population[];

    hiprandState_t state;
    hiprand_init(threadIdx.x, 0, 0, &state);

    randomInit(population + threadIdx.x, &state);
    if (threadIdx.x == 0) {
        for (int i = 0; i < N; i++) {
            printf("%d\n", (population + threadIdx.x)->path_indexes[i]);
        }
    }
}