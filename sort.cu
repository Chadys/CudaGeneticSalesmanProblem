#include "hip/hip_runtime.h"
#include "sort.cuh"
#include "header.cuh"

__device__ void swap(Individu *p, int index1, int index2){
    if (index1 == index2)
        return;
    Individu tmp = p[index1];
    p[index1] = p[index2];
    p[index2] = tmp;
}

__device__ void fusion(Individu *p, int i, int j, int endj){
    int endi = j, k = i;
    int iMoved = 0;
    while (k < endi && j < endj){
        if (p[i].score < p[j].score){
            swap(p, k, i);
            if (!iMoved)
                i++;
            else
                for (int o=i; o < i+iMoved-1; o++)
                    swap(p, o, o+1);
        } else {
            swap(p, k, j);
            if (!iMoved){
                i = j;
            }
            iMoved++;
            j++;
        }
        k++;
    }
    if (k < endi){
        fusion(p, k, i, i+iMoved);
    } else if (i < j)
        fusion(p, i, j, endj);
}

__device__ void merge_sort(Individu *population){
    int modulo = 2;
    int nbElt = 1;

    while (true){
        if (threadIdx.x % modulo == 0){
            int maxElt = threadIdx.x + nbElt * 2;
            maxElt = maxElt < blockDim.x ? maxElt : blockDim.x;
            fusion(population, threadIdx.x, threadIdx.x+nbElt, maxElt);
            nbElt = maxElt - threadIdx.x;
            if (nbElt == blockDim.x)
                return;
            modulo *= 2;
        } else {
            return;
        }
        __syncthreads();
    }
}

__device__ void bubble_sort(Individu *population){
    if((threadIdx.x % 2) == 0) {
        int even = true;
        for(int i = 0; i < blockDim.x; i++, even = !even) {
            __syncthreads();
            int tab_index = threadIdx.x + even;

            if(tab_index < blockDim.x - 1) {
                if(population[tab_index].score > population[tab_index + 1].score) {
                    swap(population, tab_index, tab_index+1);
                }
            }
        }
    }
}