#include "hip/hip_runtime.h"
#include "sort.cuh"
#include "header.cuh"

__device__ void swap(Individu *p, int index1, int index2){
    if (index1 == index2)
        return;
    Individu tmp = p[index1];
    p[index1] = p[index2];
    p[index2] = tmp;
}

__device__ void fusion(Individu *p, int i, int j, int endj){
    while (true) {
        int endi = j, k = i;
        int iMoved = 0;

        for (;k < endi && j < endj; k++) {
            if (p[i].score < p[j].score) {
                if (!iMoved)
                    i++;
                else {
                    swap(p, k, i);
                    for (int o = i; o < i + iMoved - 1; o++)
                        swap(p, o, o + 1);
                }
            } else {
                swap(p, k, j);
                if (!iMoved) {
                    i = j;
                }
                iMoved++;
                j++;
            }
        }
        if (k < endi && iMoved) {
            endj = i + iMoved;
            j = i;
            i = k;
            continue;
        }
        else if (i < j && j < endj) {
            continue;
        }
        break;
    }
}

__device__ void merge_sort(Individu *population){
    int modulo = 2;
    int nbElt = 1;

    while (true){
        if (threadIdx.x % modulo == 0){
            int maxElt = threadIdx.x + nbElt * 2;
            maxElt = maxElt < blockDim.x ? maxElt : blockDim.x;
            fusion(population, threadIdx.x, threadIdx.x+nbElt, maxElt);
            nbElt = maxElt - threadIdx.x;
            if (nbElt == blockDim.x)
                return;
            modulo *= 2;
        } else {
            return;
        }
        __syncthreads();
    }
}

__device__ void bubble_sort(Individu *population){
    if((threadIdx.x % 2) == 0) {
        int even = true;
        for(int i = 0; i < blockDim.x; i++, even = !even) {
            __syncthreads();
            int tab_index = threadIdx.x + even;

            if(tab_index < blockDim.x - 1) {
                if(population[tab_index].score > population[tab_index + 1].score) {
                    swap(population, tab_index, tab_index+1);
                }
            }
        }
    }
}